// Note this code is non-functional. Just adding snips w/ notes

// Example cuda return type
enum hipError_t { hipSuccess, hipErrorOutOfMemory };

hipError_t hipMalloc(/* void** devPtr, size_t size */) {
} // allocates device memory which is accessed via devPtr

// These are copy ops allowed
enum hipMemcpyKind {
  hipMemcpyHostToHost,
  hipMemcpyHostToDevice,
  hipMemcpyDeviceToHost,
  hipMemcpyDeviceToDevice
};

char *hipGetErrorString(hipError_t error) { // returns human readable error}

  hipError_t hipMemcpy(
      /* void* dst, const void* src, size_t count, hipMemcpyKind kind */) {
  } // xfers data from host to device

  /*
          GPU has different memory models: for now the most common
                  - Global - similar to CPU memory
                  - Shared - similar to CPU cache
  */
