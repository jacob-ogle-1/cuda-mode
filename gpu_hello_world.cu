
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel Func
__global__ void helloFromTheGPU(void) { printf("Hello from the GPU\n"); }

int main(void) {
  printf("Hello from cuda-mode!\n");
  helloFromTheGPU<<<1, 10>>>();
  hipDeviceReset();
  return 0;
}
